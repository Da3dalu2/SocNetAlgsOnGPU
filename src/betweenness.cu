/****************************************************************************
 *
 * betweenness.cpp - Serial algorithm for computing betweenness centrality
 *
 * Based on ...
 *
 * Copyright 2021 (c) 2021 by Riccardo Battistini <riccardo.battistini2(at)studio.unibo.it>
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice,
 * this list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from this
 * software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO,
 * THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS;
 * OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY,
 * WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR
 * OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF
 * ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ****************************************************************************/

//#define BENCHMARK
// TODO bandwidth measure to be tested

#include <cstdio>
#include <cstdlib>
#include "matio.h"
#include "matstorage.h"
#include "device_props.cuh"
#include "timing.cuh"
#include "errcheck.cuh"
#include "gkernels.cuh"

int main(int argc, char *argv[]) {

    matrix_pcoo_t m_coo;
    matrix_pcsr_t m_csr;
    gprops_t gp;
    int *d_row_offsets, *d_cols, *d_dist, *d_sigma;
    float *bc_gpu, *d_bc, *d_delta;
    EventTimer chrono;

    if (argc != 3) {
        fprintf(stderr, "Usage: %s [input_filename] [output_filename]", argv[0]);
        return EXIT_FAILURE;
    }

    if (get_compute_capability() != 6) {
        fprintf(stderr, "This program is meant to be executed only if compute"
                        " capability is 6.x\n");
        return EXIT_FAILURE;
    } else {
        print_gpu_overview();
    }

    /*
     * Coarse-grained parallelism.
     */
    const unsigned int threadsPerBlock = get_max_threads_per_block();
    const unsigned int blocksPerGrid = get_sm_count();
//    dim3 block = {threadsPerBlock, 1, 1};
//    dim3 grid = {blocksPerGrid, 1, 1};

    /*
     * Load Matrix Market matrix stored in .mm format as a COO matrix.
     */
    read_matrix(argv[1], &m_coo, &gp);

    /*
     * Convert the internal storage representation of the matrix from COO to
     * the more efficient CSR.
     */
    pcoo_to_pcsr(&m_coo, &m_csr);

    print_matrix_csr(&m_csr);
    printf("\n");

    /*
     * Allocate the matrix in CSR on the device.
     */
    size_t nnz = m_csr.row_offsets[m_csr.nrows];
    cudaSafeCall( hipMalloc((void**)&d_row_offsets,
                             (m_csr.nrows + 1) * sizeof(int)) );
    cudaSafeCall( hipMalloc((void**)&d_cols, nnz * sizeof(int)) );
    cudaSafeCall( hipMalloc((void**)&d_dist, m_csr.nrows * sizeof(int)) );
    cudaSafeCall( hipMalloc((void**)&d_sigma, m_csr.nrows * sizeof(int)) );
    cudaSafeCall( hipMalloc((void**)&d_delta, m_csr.nrows * sizeof(float)) );
    cudaSafeCall( hipMalloc((void**)&d_bc, m_csr.nrows * sizeof(float)) );

    /*
     * Allocate memory for the bc scores on the host.
     */
    bc_gpu = (float*) malloc(m_csr.nrows * sizeof(float));

    /*
     * Compute bc.
     */
    chrono.start();
    for(int i = 0; i < m_csr.nrows; i++) {

        int s = i;
        vtx_par_bfs<<<1, threadsPerBlock>>>(s,
                                                        d_dist,
                                                        d_sigma,
                                                        m_csr.nrows,
                                                        nnz,
                                                        d_row_offsets,
                                                        d_cols);

        cudaCheckError();

        vtx_par_dep_acc<<<1, threadsPerBlock>>>(s,
                                                            d_dist,
                                                            d_sigma,
                                                            d_delta,
                                                            d_bc,
                                                            m_csr.nrows,
                                                            nnz,
                                                            d_row_offsets,
                                                            d_cols);

        cudaCheckError();
    }

    cudaSafeCall( hipMemcpy(d_bc, bc_gpu, m_csr.nrows * sizeof(float),
                             hipMemcpyDeviceToHost));
    chrono.stop();

    /*
     * Report time elapsed and throughput.
     */
//    printf("Time elapsed: %.2f\n", chrono.elapsed());
    print_array(bc_gpu, m_csr.nrows);

#ifdef BENCHMARK
    /*
     * Compute BC with the algorithm that uses  multithreading of the BGL.
     */
    size_t nvertices = m_csr.nrows;
    bc_cpu = (float *) malloc(nvertices * sizeof(bc_cpu));

    /*
     * Check whether BC was computed correctly.
     */
    check_bc(m_csr, bc_cpu, bc_gpu);
    free(bc_cpu);
#endif

    /*
     * Cleanup.
     */
    free_matrix_pcoo(&m_coo);
    free_matrix_pcsr(&m_csr);
    cudaSafeCall( hipFree(d_row_offsets) );
    cudaSafeCall( hipFree(d_cols) );
    cudaSafeCall( hipFree(d_bc) );
    cudaSafeCall( hipFree(d_delta) );
    cudaSafeCall( hipFree(d_sigma) );
    cudaSafeCall( hipFree(d_dist) );

    close_stream(stdin);
    close_stream(stdout);
    close_stream(stderr);

    return EXIT_SUCCESS;
}
