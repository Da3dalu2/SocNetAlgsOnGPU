#include "hip/hip_runtime.h"
/****************************************************************************
 * @file bc_ep_kernel.cu
 * @author Riccardo Battistini <riccardo.battistini2(at)studio.unibo.it>
 *
 * @brief Kernel for computing Betweenness centrality on a Nvidia GPU using
 * the edge parallel technique.
 *
 * Copyright 2021 (c) 2021 by Riccardo Battistini
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice,
 * this list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from this
 * software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO,
 * THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS;
 * OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY,
 * WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR
 * OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF
 * ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 ****************************************************************************/

#include "bc_ep_kernel.cuh"

__global__ void get_vertex_betweenness_epp(double *bc,
                                           const int *rows,
                                           const int *cols,
                                           int nnz,
                                           int nvertices,
                                           int *d,
                                           unsigned long long *sigma,
                                           double *delta,
                                           int *next_source,
                                           size_t pitch_d,
                                           size_t pitch_sigma,
                                           size_t pitch_delta) {

    int tid = (int) threadIdx.x;
    if(tid >= max(2 * nnz, nvertices)) {
        return;
    }

    __shared__ int depth;
    __shared__ bool done;
    __shared__ int s;

    if (tid == 0) {
        s = (int) blockIdx.x;
    }

    int *d_row = (int *) ((char *) d + blockIdx.x * pitch_d);
    auto *delta_row = (double *) ((char *) delta + blockIdx.x * pitch_delta);
    auto *sigma_row = (unsigned long long *) ((char *) sigma +
                                              blockIdx.x * pitch_sigma);
    __syncthreads();

    /*
     * For each vertex...
     */
    while (s < nvertices) {

        if(tid == 0) {
            done = false;
            depth = 0;
        }
        __syncthreads();

        /*
         * Initialization.
         */
        for (int v = tid; v < nvertices; v += (int) blockDim.x) {
            if (v == s) {
                d_row[v] = 0;
                sigma_row[v] = 1;
            } else {
                d_row[v] = INT_MAX;
                sigma_row[v] = 0;
            }
            delta_row[v] = 0.0;
        }

        // wait for all threads to complete the initial configuration
        __syncthreads();

        /*
         * Graph traversal for shortest path discovery and counting.
         */
        while(!done) {
            __syncthreads();
            done = true;
            __syncthreads();

            /*
             * For each edge...
             */
            for(int i = tid; i < nnz; i += (int) blockDim.x) {
                int v = rows[i];

                /*
                 * If the edge is incident to a vertex in the current frontier.
                 */
                if(d_row[v] == depth) {
                    int w = cols[i];

                    if(d_row[w] == INT_MAX) {
                        d_row[w] = d_row[v] + 1;
                        done = false;
                    }

                    if(d_row[w] == (d_row[v] + 1)) {
                        atomicAdd(&sigma_row[w], sigma_row[v]);
                    }
                }
            }
            __syncthreads();

            if(tid == 0)
                depth++;
            __syncthreads();
        }

        __syncthreads();

        /*
         * Dependency accumulation by back-propagation.
         */
        while(depth > 1) {

            if(tid == 0)
                depth--;
            __syncthreads();

            /*
             * For each edge...
             */
            for(int i = tid; i < nnz; i += (int) blockDim.x) {
                int v = rows[i];

                /*
                 * If the edge is incident to a vertex in the current frontier.
                 */
                if(d_row[v] == depth) {
                    int w = cols[i];

                    if(d_row[w] == (d_row[v] + 1)) {
                        if(sigma_row[w] != 0) {
                            atomicAdd(&delta_row[v],
                                      (1.0f + delta_row[w]) *
                                      ((double) sigma_row[v] / (double) sigma_row[w]));
                        }
                    }
                }
            }
            __syncthreads();
        }

        /*
         * Compute betweenness centrality.
         */
        for (int i = tid; i < nvertices; i += (int) blockDim.x) {
            if(i != s)
                atomicAdd(&bc[i], delta_row[i]);
        }
        __syncthreads();

        if(tid == 0)
            s = atomicAdd(next_source, 1);
        __syncthreads();
    }
}

void compute_bc_gpu_epp(matrix_pcsr_t *g, double *bc, stats_t *stats) {

    double tstart, tend, first_tstart, last_tend;

    first_tstart = get_time();
    int nnz = (g->row_offsets[g->nrows]);

    unsigned long long *d_sigma;
    double *d_bc, *d_delta;
    int *d_rows, *d_cols, *d_dist, *d_next_source;
    size_t pitch_d, pitch_sigma, pitch_delta;

    auto rows = (int *) malloc(nnz * sizeof(int));
    expand_row_pointer(g->nrows, g->row_offsets, rows);

    /*
     * Setup block and grid dimensions.
     */
    const unsigned int sm_count = get_sm_count();
    int next_source = (int) sm_count;

    const unsigned int threads_per_block = get_max_threads_per_block();
    const unsigned int blocks_per_grid = sm_count;
    dim3 block = {threads_per_block, 1, 1};
    dim3 grid = {blocks_per_grid, 1, 1};

    /*
    * Load the COO matrix on the device.
    */
    cudaSafeCall(hipMalloc((void **) &d_rows,
                            nnz * sizeof(int)));
    cudaSafeCall(hipMalloc((void **) &d_cols,
                            nnz * sizeof(int)));

    cudaSafeCall(hipMemcpy(d_rows, rows,
                            nnz * sizeof(int),
                            hipMemcpyHostToDevice));
    cudaSafeCall(hipMemcpy(d_cols, g->cols,
                            nnz * sizeof(int),
                            hipMemcpyHostToDevice));

    free(rows);

    /*
    * Load bc.
    */
    cudaSafeCall(hipMalloc((void **) &d_bc, g->nrows * sizeof(double)));
    cudaSafeCall(hipMemset(d_bc, 0, g->nrows * sizeof(double)));

    /*
     * Load auxiliary arrays for bc.
     */
    cudaSafeCall(hipMallocPitch((void **) &d_dist, &pitch_d,
                                 g->nrows * sizeof(int), grid.x));
    cudaSafeCall(hipMallocPitch((void **) &d_sigma, &pitch_sigma,
                                 g->nrows * sizeof(unsigned long long),
                                 grid.x));
    cudaSafeCall(hipMallocPitch((void **) &d_delta, &pitch_delta,
                                 g->nrows * sizeof(double), grid.x));

    /*
     * Load single-variables.
     */
    cudaSafeCall(hipMalloc((void **) &d_next_source, sizeof(int)));
    cudaSafeCall(hipMemcpy(d_next_source, &next_source,
                            sizeof(int),
                            hipMemcpyHostToDevice));

    tend = get_time();
    stats->load_time = tend - first_tstart;

    /*
     * Execute the bc computation.
     */
    tstart = get_time();
    get_vertex_betweenness_epp<<<grid, block>>>(d_bc,
                                                d_rows,
                                                d_cols,
                                                nnz,
                                                g->nrows,
                                                d_dist,
                                                d_sigma,
                                                d_delta,
                                                d_next_source,
                                                pitch_d,
                                                pitch_sigma,
                                                pitch_delta);

    cudaCheckError();

    cudaSafeCall(hipMemcpy(bc, d_bc,
                            g->nrows * sizeof(double),
                            hipMemcpyDeviceToHost));

    /*
     * Count each edge only one time.
     */
    for (int k = 0; k < g->nrows; k++)
        bc[k] /= 2;

    cudaSafeCall(hipDeviceSynchronize());
    tend = get_time();
    stats->bc_comp_time = tend - tstart;
    tstart = get_time();

    /*
     * Device resource deallocation.
     */
    cudaSafeCall(hipFree(d_rows));
    cudaSafeCall(hipFree(d_next_source));
    cudaSafeCall(hipFree(d_cols));
    cudaSafeCall(hipFree(d_bc));
    cudaSafeCall(hipFree(d_sigma));
    cudaSafeCall(hipFree(d_dist));
    cudaSafeCall(hipFree(d_delta));

    last_tend = get_time();
    stats->unload_time = last_tend - tstart;
    stats->total_time = last_tend - first_tstart;
}
